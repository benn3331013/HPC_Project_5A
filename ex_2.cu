
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 1024
#define d 1024

int N_Blocks = N/d+10;
int NTPB = 1024;

void testCUDA(hipError_t error, const char *file, int line)  {
	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

void QuickSort( int* array , size_t end) {
    if (end > 1) {
        int pivot = array[end-1];

        size_t cursor = 0;
        int swap_variable;
        for (size_t i = 0; i < end - 1 ; i++) {
            if ( array[i] < pivot ) {
                swap_variable = array[i];
                array[i] = array[cursor];
                array[cursor] = swap_variable;

                cursor = cursor + 1;
            }
        }
        swap_variable = array[end-1];
        array[end-1] = array[cursor];
        array[cursor] = swap_variable;

        QuickSort(array , cursor );
        QuickSort(array + cursor + 1, end - cursor - 1);
    }
}

bool IsSortedAscending(int* array, size_t length) {
    for(size_t i=1; i<length; i++) {
        if (array[i-1]>array[i]) {
            return false;
        }
    }
    return true;
}

bool IsSortedDescending(int* array, size_t length) {
    for(size_t i=1; i<length; i++) {
        if (array[i-1]<array[i]) {
            return false;
        }
    }
    return true;
}

/*
    * @brief Generates a random array
    * @param arr the array that'll be malloced and filled
    * @param length The size of the array
    */
void GenerateRandomArray( int** arr, const size_t length ) {
    *arr = (int*) malloc(length*sizeof(int));

    for (size_t i=0 ; i<length ; i++) {
        (*arr)[i] = rand()%(5*length);
    }
    QuickSort(*arr, length);
}

void GenerateUnsortedRandomArray( int** arr, const size_t length ) {
    *arr = (int*) malloc(length*sizeof(int));

    for (size_t i=0 ; i<length ; i++) {
        (*arr)[i] = rand()%(5*length);
    }
}

void PrintList(int* A, size_t length){
    
    for (int i = 0; i < length; i++) {
        printf("%d ", A[i]);
    }
    printf("\n");
}



__global__ void mergeSmallBatch_k(const int** A, const int** B, int** M, const size_t* NA, const size_t* NB) {
//  Since the number of threads per block is a multiple of d, that means a single block can merge several arrays

//  This is the thread's position in the array
    int tidx = threadIdx.x%d;
//  This is the index of the array from the arrays given to a specific block
    int Qt = (threadIdx.x-tidx)/d;
//  This is the "global" index of the array, in which the thread is going to work
    int gbx = Qt + blockIdx.x*(blockDim.x/d);

//  the thread works on M[gbx][tidx]

    if ( gbx >= N ) {// excedent block
        return;
    }

    int K[2];
    int P[2];

    if (tidx > NA[gbx]) {
        K[0] = tidx - NA[gbx];
        K[1] = NA[gbx];

        P[0] = NA[gbx];
        P[1] = tidx - NA[gbx];
    } else {
        K[0] = 0;
        K[1] = tidx;

        P[0] = tidx;
        P[1] = 0;
    }

    while (true) {
        int offset = abs((K[1] - P[1]) / 2);
        int Q[] = {K[0] + offset, K[1] - offset};

        if (Q[1] >= 0 && Q[0] <= NB[gbx] && (Q[1] == NA[gbx] || Q[0] == 0 || A[gbx][Q[1]] > B[gbx][Q[0] - 1])) {
            if (Q[0] == NB[gbx] || Q[1] == 0 || A[gbx][Q[1] - 1] <= B[gbx][Q[0]]) {
                if (Q[1] < NA[gbx] && (Q[0] == NB[gbx] || A[gbx][Q[1]] <= B[gbx][Q[0]]))
                    M[gbx][tidx] = A[gbx][Q[1]];
                else
                    M[gbx][tidx] = B[gbx][Q[0]];
                break;
            } else {
                K[0] = Q[0] + 1;
                K[1] = Q[1] - 1;
            }
        } else {
            P[0] = Q[0] - 1;
            P[1] = Q[1] + 1;
        }
    }
}

__global__ void SortSmall_k(int **M, const size_t *NM, int j, int k){
// Each block sorts one array


//  This is the thread's position in the array
    int tidx = threadIdx.x;

// This is the id of the table that the thread is going to help sorting
    int bid = blockIdx.x;


// HERE DUNNO WHY THIS IS NOT NECESSARY
//    if (tidx >= NM[bid] || k >= NM[bid])
    //if (tidx >= NM[bid])
    //    return;


    int ixj = tidx^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>tidx) {
        if ((tidx&k)==0) {
        /* Sort ascending */
        if (M[bid][tidx]>M[bid][ixj]) {
            /* exchange(i,ixj); */
            int temp = M[bid][tidx];
            M[bid][tidx] = M[bid][ixj];
            M[bid][ixj] = temp;
        }
        }
        if ((tidx&k)!=0) {
        /* Sort descending */
        if (M[bid][tidx]<M[bid][ixj]) {
            /* exchange(i,ixj); */
            int temp = M[bid][tidx];
            M[bid][tidx] = M[bid][ixj];
            M[bid][ixj] = temp;
        }
        }
    }

}


void SortSmall(int **M, const size_t* NM)
{
//Sorts a group of small arrays using GPU parallelized Bitonic Sort

    printf("Preparing Sort...\n");

    int **M_GPU;
    size_t *NM_GPU;
    //size_t size = NUM_VALS * sizeof(float);

    testCUDA(hipMalloc(&NM_GPU, N * sizeof(size_t)));

    int** tempo_array;
    tempo_array = (int**) malloc(N*sizeof(int*));
    
    
    printf("Creating M_GPU...\n");

    for (int i = 0; i < N; i++){
        testCUDA(hipMalloc(&tempo_array[i], NM[i] * sizeof(int)));
    }


    testCUDA(hipMalloc(&M_GPU, N * sizeof(int*)));
    
    for (int i = 0; i < N; i++){
        testCUDA(hipMemcpy(tempo_array[i], M[i], NM[i] * sizeof(int), hipMemcpyHostToDevice));
    }
    
    testCUDA(hipMemcpy(M_GPU, tempo_array, N*sizeof(int*), hipMemcpyHostToDevice));

    printf("Done Preparing Sort!\n");


    printf("Begin Sorting Procedure...\t");

    int j, k;
    /* Major step */
    for (k = 2; k <= d; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            //printf("k : %d || j : %d\n", k, j);
            //SortSmall_k<<<N_Blocks, NTPB>>>(M_GPU, NM_GPU, j, k);
            SortSmall_k<<<N, NTPB>>>(M_GPU, NM_GPU, j, k);
            //printf("\n");
        }
    }

    printf("Done Sorting!\n");
    
    printf("Importing M from GPU...\t");

    for (size_t i=0;i<N;i++) {
        testCUDA(hipMemcpy(M[i], tempo_array[i], d * sizeof(int), hipMemcpyDeviceToHost));
    }

    printf("M successfully imported from GPU\n");

//TODO
/*  
    for (int i = 0; i < N; i++){
        //testCUDA(cudaFree(tempo_array+i));
        testCUDA(cudaFree(M_GPU[i]));
    }
    puts("Check");

    free(tempo_array);
    //free(tempo);
    testCUDA(cudaFree(NM_GPU));
    testCUDA(cudaFree(M_GPU));
*/
}


int main()
{
    printf("Creating arrays...\t");
    srand(time(NULL));
    int a_sizes[] = {1, 4, 6};

    int** A = (int**) malloc(N*sizeof(int*));
    size_t* NA = (size_t*) malloc(N*sizeof(size_t));
    for (size_t i=0;i<N;i++) {
        NA[i] = a_sizes[i%3];
        GenerateRandomArray(A+i, NA[i]);
    }


    int** B = (int**) malloc(N*sizeof(int*));
    size_t* NB = (size_t*) malloc(N*sizeof(size_t));
    for (size_t i=0;i<N;i++) {
        NB[i] = d - NA[i];
        GenerateRandomArray(B+i, NB[i]);
    }


    int** S = (int**) malloc(N*sizeof(int*));
    size_t* NS = (size_t*) malloc(N*sizeof(size_t));
    for (size_t i=0;i<N;i++) {
        NS[i] = d;
        GenerateUnsortedRandomArray(S+i, NS[i]);
    }
    printf(" Done!\n");


//  We could create contiguous arrays...
    printf("Creating GPU arrays...\t");
    int** A_GPU;
    int** B_GPU;
    int** M_GPU;

    size_t* NA_GPU;
    size_t* NB_GPU;

    testCUDA(hipMalloc(&NA_GPU, N * sizeof(size_t)));
    testCUDA(hipMalloc(&NB_GPU, N * sizeof(size_t)));


    int** tempo_array;
    tempo_array = (int**) malloc(N*sizeof(int*));

//  Creating M_GPU
    printf("Creating M_GPU...\t");
    testCUDA(hipMalloc(&M_GPU, N * sizeof(int*)));
    for (size_t i=0;i<N;i++) {
        testCUDA(hipMalloc(tempo_array+i, (NB[i]+NA[i]) * sizeof(int)));
    }
    testCUDA(hipMemcpy(M_GPU, tempo_array, N * sizeof(int*), hipMemcpyHostToDevice));
    printf("Done!\n");

//  Creating A_GPU
    printf("Creating A_GPU...\t");
    testCUDA(hipMalloc(&A_GPU, N * sizeof(int*)));
    for (size_t i=0;i<N;i++) {
        testCUDA(hipMalloc(tempo_array+i, NA[i] * sizeof(int)));
        testCUDA(hipMemcpy(tempo_array[i], A[i], NA[i] * sizeof(int), hipMemcpyHostToDevice));
    }
    testCUDA(hipMemcpy(A_GPU, tempo_array, N * sizeof(int*), hipMemcpyHostToDevice));
    printf("Done!\n");

//  Creating B_GPU
    printf("Creating B_GPU...\t");
    testCUDA(hipMalloc(&B_GPU, N * sizeof(int*)));
    for (size_t i=0;i<N;i++) {
        testCUDA(hipMalloc(tempo_array+i, NB[i] * sizeof(int)));
        testCUDA(hipMemcpy(tempo_array[i], B[i], NB[i] * sizeof(int), hipMemcpyHostToDevice));
    }
    testCUDA(hipMemcpy(B_GPU, tempo_array, N * sizeof(int*), hipMemcpyHostToDevice));
    printf("Done!\n");

//  Filling the arrays 
    printf("Copying the sizes...\t");
    testCUDA(hipMemcpy(NA_GPU, NA, N * sizeof(size_t), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(NB_GPU, NB, N * sizeof(size_t), hipMemcpyHostToDevice));
    printf(" Done!\n");


    
    printf("Merging...\t");
    mergeSmallBatch_k<<<N_Blocks, NTPB>>>((const int**)A_GPU, (const int**)B_GPU, M_GPU, NA_GPU, NB_GPU);
    printf("Done!\n");
    

    int** M = (int**) malloc(N * sizeof(int*));
    int** tempo = (int**) malloc(N * sizeof(int*));
    testCUDA(hipMemcpy(tempo, M_GPU, N * sizeof(int*), hipMemcpyDeviceToHost));
    printf("M created\n");
    for (size_t i=0;i<N;i++) {
        M[i] = (int*) malloc(d * sizeof(int));
        testCUDA(hipMemcpy(M[i], tempo[i], d * sizeof(int), hipMemcpyDeviceToHost));
    }
    printf("M filled\n");



//    PrintList(M, (NA + NB));
    printf("Verifying Merge result\n");
    for (size_t i=0;i<N;i++) {  
        if (! IsSortedAscending(M[i],d) ) {
            printf("The result isn't correct...\n");
            exit(EXIT_FAILURE);
        }
    }
    printf("The result is correct!\n");


    SortSmall(S, NS);

    printf("Verifying Sort result\n");
    for (size_t i=0;i<N;i++) {  
        if (! IsSortedAscending(S[i],d) ) {
            printf("The result isn't correct..., i = %zd\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("The result is correct!\n");


    testCUDA(hipFree(A_GPU));
    testCUDA(hipFree(B_GPU));
    testCUDA(hipFree(M_GPU));
    

    for (size_t i=0;i<N;i++) {
        free(A[i]);
        free(B[i]);
        free(M[i]);
        free(S[i]);
    }
    free(A);
    free(B);
    free(M);
    free(S);

    return 0;
}